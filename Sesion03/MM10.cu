
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 16

#ifndef PINNED
#define PINNED 0
#endif


// Matriz por Matriz
// C(NxM) <- A(NxP) * B (PxM)

__global__ void Kernel01 (int N, int M, int P, float *A, float *B, float *C) {
	__shared__ float sA[SIZE][SIZE];
	__shared__ float sB[SIZE][SIZE];

	int bx = blockIdx.x; int by = blockIdx.y;
        int tx = threadIdx.x; int ty = threadIdx.y;
        int row = by * SIZE + ty;
        int col = bx * SIZE + tx;

	float tmp = 0.0;
	for (int m=0; m< (P/SIZE); m++) {
		sA[ty][tx] = A[row*P + m*SIZE + tx];
		sB[ty][tx] = B[col + (m*SIZE + ty)*M];
		__syncthreads();
		for (int k=0; k<SIZE; k++)
			tmp += sA[ty][k] * sB[k][tx];
		__syncthreads();
	}
	C[row*M+col] = tmp;	
}
/*__global__ vois MMkernel2 (float *dA, float *dB, float *dC, int N) {

	__shared__ float sA[size][size];
	__shared__ float sB[size][size];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int row = by * size + ty;
	int col = bx * size + tx;

	float tmp = 0.0;
	for (m=0; m< (N/size); m++) {
		sA[ty][tx] = dA[row*N + m*size + tx];
		sB[ty][tx] = dB[col + (m*size + ty)*N];
		_syncthreads();
		for (int k=0; k<size; k++)
			tmp += sA [ty][k] * sB[k][tx];
		_syncthreads();
	}
	dC[row*N+col] = tmp;
}*/


void InitM(int N, int M, float *Mat);
int TestMM(int N, int M, int P, float *A, float *B, float *C);


// Invocacion:
// ./ejecutable TAM test
// TAM es el la dimension de las matrices
// test == 'Y', comprueba que el resultado sea correcto
// test == 'N', NO comprueba que el resultado (Util para tomar tiempos)
// Por defecto, N = 2048, test == 'N'

int main(int argc, char** argv)
{
  unsigned int N, M, P;
  unsigned int numBytesM1, numBytesM2, numBytesRes;
  unsigned int nBlocks, mBlocks, nThreads;
 
  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;

  char test;

  // Dimension de las matrices NxN y comprobacion resultado
  if (argc == 1)      { test = 'N'; N = 2048; M = 2048; P = 2048; }
  else if (argc == 2) { test = 'N'; N = atoi(argv[1]); M = N; P = N; }
  else if (argc == 3) { test = *argv[2]; N = atoi(argv[1]);M = N; P = N; }
  else if (argc == 4) { test = 'N'; N = atoi(argv[1]); P = atoi(argv[2]); M = atoi(argv[3]);}
  else if (argc == 5) { test = *argv[4]; N = atoi(argv[1]); P = atoi(argv[2]); M = atoi(argv[3]);}
  else { printf("Usage: ./exe TAM1x TAM1y/2x TAM2y test\n"); exit(0); }

  // numero de Threads en cada dimension 
  nThreads = SIZE;

  // numero de Blocks en cada dimension 
  nBlocks = N/nThreads; 
  mBlocks = M/nThreads;

  printf("%d:%d, %d:%d\n", N, nBlocks, M, mBlocks);
  
  
  numBytesM1 = N * P * sizeof(float);
  numBytesM2 = M * P * sizeof(float);
  numBytesRes = M * N * sizeof(float);

  dim3 dimGrid(nBlocks, mBlocks, 1);
  dim3 dimBlock(nThreads, nThreads, 1);

  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  if (PINNED) {
    // Obtiene Memoria [pinned] en el host
    hipHostMalloc((float**)&h_A, numBytesM1, hipHostMallocDefault); 
    hipHostMalloc((float**)&h_B, numBytesM2, hipHostMallocDefault); 
    hipHostMalloc((float**)&h_C, numBytesRes, hipHostMallocDefault); 
  }
  else {
    // Obtener Memoria en el host
    h_A = (float*) malloc(numBytesM1); 
    h_B = (float*) malloc(numBytesM2); 
    h_C = (float*) malloc(numBytesRes); 
  }

  // Inicializa las matrices
  InitM(N, P, h_A);
  InitM(P, M, h_B);

  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
  
  // Obtener Memoria en el device
  hipMalloc((float**)&d_A, numBytesM1); 
  hipMalloc((float**)&d_B, numBytesM2); 
  hipMalloc((float**)&d_C, numBytesRes); 

  // Copiar datos desde el host en el device 
  hipMemcpy(d_A, h_A, numBytesM1, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, numBytesM2, hipMemcpyHostToDevice);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);
  
  // Ejecutar el kernel 
  Kernel01<<<dimGrid, dimBlock>>>(N, P, M, d_A, d_B, d_C);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host 
  hipMemcpy(h_C, d_C, numBytesRes, hipMemcpyDeviceToHost); 

  // Liberar Memoria del device 
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nKERNEL 01\n");
  printf("Dimensiones: %dx%d\n", N, N);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
  if (PINNED) printf("Usando Pinned Memory\n");
         else printf("NO usa Pinned Memory\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
  printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
  printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

  if (test == 'N')
    printf ("NO TEST\n");
  else  if (TestMM(N, N, N, h_A, h_B, h_C))
    printf ("TEST PASS\n");
  else
    printf ("TEST FAIL\n");

  if (PINNED) {
    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
  }
  else {
    free(h_A); free(h_B); free(h_C);
  }

}
