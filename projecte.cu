#include "hip/hip_runtime.h"
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <vector>
#include <ctime>
#include "readpng.cpp"

typedef float*  vectorr;

#define SIZE 32

#ifndef PINNED
#define PINNED 0
#endif

__global__ void applyFilt(int N, int M, int P, vectorr entrada, vectorr filt, vectorr sortida) {
	/* N ample de la matriu
	 * M alçada de la matriu
	 * P mida filtre
	 * entrada matriu d'entrada
	 * sortida matriu de sortida
	 * filt matriu filtre
	 */
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int mod = (P-1)/2;
	int j, k, l;
	float acc;
	j = column;
//or (j = mod; j < M-mod; ++j)
//
	printf("row: %d, column %d\n", row, column);
	if (row >= mod && column >= mod && row < N-mod && column < M-mod) 
	{
		acc = 0;
		printf("AUX %d\n", filt[0]); 
		for (k=0; k<P; k++) {
			//calculem l'index dintre del filtre utilitzant 
			//mod com a pivot
			int indexi = (row+(k-mod))*N+j-mod;
			for (l=0; l<P; l++) {
				float aux = entrada[indexi+l];
				float auxb = filt[k*P+l];
				acc = acc + aux * auxb;
			}
			printf("AUX %d", filt[k*P+0]); 
		}
		sortida[row*N+j] =  acc;
	}
	else if (row >= 0 && column >= 0 && row < N-1 && column < M-1){
		acc = entrada[row*N+j];
		printf("no he entrat. %d, %d\n", row, column);
	}
	else {
		printf("no he entrat. %d, %d\n", row, column);
	}
//
}

vectorr img2bw(int N, int M, unsigned char** foto)
{
			//0.21 R + 0.72 G + 0.07 B
	int i, j;
	//~ int count=0;
	int size = N*M;
	vectorr ret = (vectorr) malloc(size*sizeof(float));
	//~ fprintf(stderr, "%d -> %lu\n", (N*M), (N*M*sizeof(float)));
	for (i=0; i<M; i++) {
		unsigned char* r = foto[i];			
		for (j=0; j<N; j++) {
			unsigned char* ptr = &(r[j*4]);
			float aux = float(0.21*ptr[0] + 0.72*ptr[1] + 0.07*ptr[2])/255.0;
			ret[i*M+j] = aux;
		}
	}
	//~ fprintf(stderr, "Run over i: %d, j: %d, : count: %d", i, j, count);
	return ret;
}

vectorr gaussFilt(int N)
{
	vectorr ret = (vectorr) malloc(N*N*sizeof(float));
	float value = 1.0/(N*N);
	for (int i = 0; i < N*N; i++)
		ret[i] = value;
	return ret;
}

vectorr laplaceFilt()
{
	
	vectorr ret = (vectorr) malloc(3*3*sizeof(float));
	ret[0] = 0;
	ret[1] = -4;
	ret[2] = 0;
	ret[3] = -4;
	ret[4] = 16;
	ret[5] = -4;
	ret[6] = 0;
	ret[7] = -4;
	ret[8] = 0;
	return ret;
}


void print(int N, int M, float *C)
{
	int i, j;
	for (i=0; i<N; i++) {
		for (j=0; j<M; j++) {
			printf("%f ", C[i*M+j]);
		}
		printf("\n");
	}
}



unsigned char** greyChar(int N, int M, vectorr m)
{
	unsigned char** ret = (unsigned char**) malloc(M*sizeof(unsigned char*));
	//~ std::vector<bool> v(N*M*sizeof(float));
	int i,j;
	//int count = 0;	
	for (i=0; i<M; i++) {
		unsigned char* row = (unsigned char*) malloc(4*N*sizeof(unsigned char));
		for (j=0; j<N; j++) {
			unsigned char c = m[i*M+j]*255;
			//~ if (v[i*N+j])
				//~ count++;
			//~ else
				//~ v[i*N+j] = true;
			int k = j*4;
			row[k+0] = c;
			row[k+1] = c;
			row[k+2] = c;
			row[k+3] = (unsigned char) 255;
		}
		ret[i] = row;
	}
	//~ fprintf(stderr, "\nCOUNT: %d\n", count);
	return ret;
}



void printPerf(char *nombre, int N,unsigned int nThreads,unsigned int  nBlocks,
		float TiempoTotal, float TiempoKernel, float ops) {
	float GFlop = ops;
	float totalSec = TiempoTotal/1000.0;
	float kernelSec = TiempoKernel/1000.0;
	printf("\nFilter CUDA: %s\n", nombre);
	printf("Dimensiones: %dx%d\n", N, N);
	printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
	printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
	printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
	printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
	printf("Rendimiento Global: %4.2f GFLOPS\n", (GFlop / totalSec));
	printf("Rendimiento Kernel: %4.2f GFLOPS\n\n", (GFlop / kernelSec));
}

int main(int argc, char** argv)
{
	int c;
	int gauss=0, laplace=0, sharpen=0, bumping=0, noise=0, histo=0;
	char *image = NULL;
	char *output = NULL;
	clock_t begin, end;
	
	unsigned int N;
	unsigned int numBytes, numBytesF;
	unsigned int nBlocks, nThreads;
	
	vectorr 	h_in, d_in;
	vectorr		h_out_gauss, h_out_lapla, d_out;
	vectorr		h_filt_gauss, d_filt_gauss,
				h_filt_lapla, d_filt_lapla;
	
	float TiempoTotal, TiempoKernel,
			elapsed_read,
			elapsed_write;
	hipEvent_t E0, E1, E2, E3;

	
	while ((c = getopt (argc, argv, "g:lsbnaHi:o:")) != -1)
	{
		switch (c)
		{
			case 'a':
				gauss=3;
				laplace=sharpen=bumping=noise=histo=1;
				break;
			case 'b':
				bumping = 1;
				break;
			case 'g':
				gauss=atoi(optarg);
				break;
			case 'H':
				histo = 1;
				break;
			case 'i':
				image = optarg;
				break;
			case 'l':
				laplace = 1;
				break;
			case 'n':
				noise = 1;
				break;
			case 'o':
				output = optarg;
				break;
			case 's':
				sharpen = 1;
				break;
			default:
				abort();
		}
	}
	if (image == NULL) {
		fprintf(stderr, "ERROR: Necesito una imagen\n");
		return -1;
	}
	if (output == NULL) {
		fprintf(stderr, "WARN: Tomando salida por defecto : salida.png\n");
		output = "salida.png";
	}
	
	//~ fprintf(stderr, "toread\n");
	begin = clock();
	read_png_file(image);
	h_in = img2bw(width, height, row_pointers);
	end = clock();
	elapsed_read = (float(end-begin)/CLOCKS_PER_SEC);
	if (png_get_color_type(png_ptr, info_ptr) == PNG_COLOR_TYPE_RGB)
		abort_("[process_file] input file is PNG_COLOR_TYPE_RGB but must be PNG_COLOR_TYPE_RGBA ",
						"(lacks the alpha channel)");

	if (png_get_color_type(png_ptr, info_ptr) != PNG_COLOR_TYPE_RGBA)
		abort_("[process_file] color_type of input file must be PNG_COLOR_TYPE_RGBA (%d) (is %d)",
					 PNG_COLOR_TYPE_RGBA, png_get_color_type(png_ptr, info_ptr));
	
	//~ printf("Filtre de gauss\n");
	//~ print(gauss,gauss, h_filt);

	N = height;
	  // numero de Threads en cada dimension 
	nThreads = SIZE;

  // numero de Blocks en cada dimension 
	//nBlocks = N/nThreads; 
	nBlocks = (N+nThreads-1)/nThreads;
	numBytes = N * N * sizeof(float);
	numBytesF = gauss*gauss * sizeof(float);
	
	dim3 dimGrid(nBlocks, nBlocks, 1);
	dim3 dimBlock(nThreads, nThreads, 1);
	
	if (gauss != 0)
	{
		h_filt_gauss = gaussFilt(gauss);
		int mod = (gauss-1)/2; //margen que no se calculará
		hipEventCreate(&E0);
		hipEventCreate(&E1);
		hipEventCreate(&E2);
		hipEventCreate(&E3);
		
		// Obtener Memoria en el host
		//~ row_pointers = (unsigned char**) malloc(numBytes); 
		h_out_gauss = (vectorr) malloc(numBytes); 
	  
		hipEventRecord(E0, 0);
		hipEventSynchronize(E0);


		// Obtener Memoria en el device
		//~ hipMalloc((unsigned char **)&d_in, numBytes); 
		hipMalloc((vectorr*)&d_in, numBytes); 
		hipMalloc((vectorr*)&d_out, numBytes); 
		hipMalloc((vectorr*)&d_filt_gauss, numBytesF); 

		// Copiar datos desde el host en el device 
		hipMemcpy(d_in, h_in, numBytes, hipMemcpyHostToDevice);
		hipMemcpy(d_out, h_out_gauss, numBytes, hipMemcpyHostToDevice);
		hipMemcpy(d_filt_gauss, h_filt_gauss, numBytesF, hipMemcpyHostToDevice);

		hipEventRecord(E1, 0);
		hipEventSynchronize(E1);

		//~ fprintf(stderr, "Abans kernel\n");
		// Ejecutar el kernel 
		/***********************/
		/***********************/
		/***********************/
		/***********************/
		//~ print(width, height, h_in);
		//~ print(gauss, gauss, h_filt);
		applyFilt<<<dimGrid, dimBlock>>>(N, N, gauss, d_in, d_filt_gauss, d_out);
		/***********************/
		/***********************/
		/***********************/
		/***********************/
		hipEventRecord(E2, 0);
		hipEventSynchronize(E2);

		// Obtener el resultado desde el host 
		hipMemcpy(h_out_gauss, d_out, numBytes, hipMemcpyDeviceToHost); 

		// Liberar Memoria del device 
		hipFree(d_in);
		hipFree(d_filt_gauss);
		hipFree(d_out);

		hipEventRecord(E3, 0);
		hipEventSynchronize(E3);

		hipEventElapsedTime(&TiempoTotal,  E0, E3);
		hipEventElapsedTime(&TiempoKernel, E1, E2);

		begin = clock();
		row_pointers = greyChar(width, height, h_out_gauss);
		write_png_file(output);	
		end = clock();
		elapsed_write = (float(end-begin)/CLOCKS_PER_SEC);
		
		
		float ops = ((N-gauss)/1000000000.0)*(N-gauss)*(gauss)*(gauss)*2.0;
		printf("OPS: %f", ops);
		printPerf("Gauss", N, nThreads, nBlocks,
				TiempoTotal+elapsed_read+elapsed_write,
				TiempoKernel, ops);
		
		hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
	}
	if (laplace)
	{
		h_filt_lapla = laplaceFilt();
		
		hipEventCreate(&E0);
		hipEventCreate(&E1);
		hipEventCreate(&E2);
		hipEventCreate(&E3);
		
		// Obtener Memoria en el host
		//~ row_pointers = (unsigned char**) malloc(numBytes); 
		h_out_lapla = (vectorr) malloc(numBytes); 
	  
		hipEventRecord(E0, 0);
		hipEventSynchronize(E0);


		// Obtener Memoria en el device
		//~ hipMalloc((unsigned char **)&d_in, numBytes); 
		hipMalloc((vectorr*)&d_in, numBytes); 
		hipMalloc((vectorr*)&d_out, numBytes); 
		hipMalloc((vectorr*)&d_filt_lapla, numBytesF); 

		// Copiar datos desde el host en el device 
		hipMemcpy(d_in, h_in, numBytes, hipMemcpyHostToDevice);
		hipMemcpy(d_out, h_out_lapla, numBytes, hipMemcpyHostToDevice);
		hipMemcpy(d_filt_lapla, h_filt_lapla, numBytesF, hipMemcpyHostToDevice);

		hipEventRecord(E1, 0);
		hipEventSynchronize(E1);

		//~ fprintf(stderr, "Abans kernel\n");
		// Ejecutar el kernel 
		/***********************/
		/***********************/
		/***********************/
		/***********************/
		//~ print(width, height, h_in);
		print(3, 3, h_filt_lapla);
		applyFilt<<<dimGrid, dimBlock>>>(N, N, 3, d_in, d_filt_lapla, d_out);
		hipDeviceSynchronize();
		/***********************/
		/***********************/
		/***********************/
		/***********************/
		hipEventRecord(E2, 0);
		hipEventSynchronize(E2);

		// Obtener el resultado desde el host 
		hipMemcpy(h_out_lapla, d_out, numBytes, hipMemcpyDeviceToHost); 

		// Liberar Memoria del device 
		hipFree(d_in);
		hipFree(d_filt_lapla);
		hipFree(d_out);

		hipEventRecord(E3, 0);
		hipEventSynchronize(E3);

		hipEventElapsedTime(&TiempoTotal,  E0, E3);
		hipEventElapsedTime(&TiempoKernel, E1, E2);

		begin = clock();
		row_pointers = greyChar(width, height, h_out_lapla);
		fprintf(stderr, "GREYED\n");
		write_png_file(output);	
		end = clock();
		elapsed_write = (float(end-begin)/CLOCKS_PER_SEC);
		
		
		float ops = ((N-3)/1000000000.0)*(N-3)*(gauss)*(gauss)*2.0;
		printPerf("LAPLACE", N, nThreads, nBlocks,
				TiempoTotal+elapsed_read+elapsed_write,
				TiempoKernel, ops);
		
		hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
	}
	return 0;
}    

