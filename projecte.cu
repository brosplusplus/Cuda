#include "hip/hip_runtime.h"
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <vector>
#include <ctime>
#include "readpng.cpp"

typedef float**  vectorr;

#define SIZE 16

#ifndef PINNED
#define PINNED 0
#endif

__global__ void img2bw(int N, int M, unsigned char* entrada, vectorr sortida) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned char *r = entrada[row];
	unsigned char *ptr = &(r[col*4]);
	float aux = float(0.21*ptr[0] + 0.72*ptr[1] + 0.07*ptr[2])/255.0;
	sortida[row][col] = aux; 
}


void print(int N, int M, float *C)
{
	int i, j;
	for (i=0; i<N; i++) {
		for (j=0; j<M; j++) {
			printf("%f ", C[i*M+j]);
		}
		printf("\n");
	}
}



unsigned char** greyChar(int N, int M, vectorr m)
{
	unsigned char** ret = (unsigned char**) malloc(M*sizeof(unsigned char*));
	std::vector<bool> v(N*M*sizeof(float));
	int i,j;
	int count = 0;
	for (i=0; i<M; i++) {
		unsigned char* row = (unsigned char*) malloc(4*N*sizeof(unsigned char));
		for (j=0; j<N; j++) {
			unsigned char c = m[i][j]*255;
			if (v[i*N+j])
				count++;
			else
				v[i*N+j] = true;
			int k = j*4;
			row[k+0] = c;
			row[k+1] = c;
			row[k+2] = c;
			row[k+3] = (unsigned char) 255;
		}
		ret[i] = row;
	}
	fprintf(stderr, "\nCOUNT: %d\n", count);
	return ret;
}

int main(int argc, char** argv)
{
	int c;
	int gauss=0, laplace=0, sharpen=0, bumping=0, noise=0, histo=0;
	char *image = NULL;
	char *output = NULL;
	clock_t begin, end;
	
	unsigned int N;
	unsigned int numBytes, numBytesF;
	unsigned int nBlocks, nThreads;
	
	unsigned char** 	h_in, d_in;
	vectorr			h_out, d_out;
	
	float TiempoTotal, TiempoKernel;
	hipEvent_t E0, E1, E2, E3;

	
	while ((c = getopt (argc, argv, "glsbnaHi:o:")) != -1)
	{
		switch (c)
		{
			case 'a':
				gauss=laplace=sharpen=bumping=noise=histo=1;
				break;
			case 'b':
				bumping = 1;
				break;
			case 'g':
				gauss=1;
				break;
			case 'H':
				histo = 1;
				break;
			case 'i':
				image = optarg;
				break;
			case 'l':
				laplace = 1;
				break;
			case 'n':
				noise = 1;
				break;
			case 'o':
				output = optarg;
				break;
			case 's':
				sharpen = 1;
				break;
			default:
				abort();
		}
	}
	if (image == NULL) {
		fprintf(stderr, "ERROR: Necesito una imagen\n");
		return -1;
	}
	if (output == NULL) {
		fprintf(stderr, "WARN: Tomando salida por defecto : salida.png\n");
		output = "salida.png";
	}
	fprintf(stderr, "toread\n");
	begin = clock();
	read_png_file(image);
	end = clock();
	//~ double elapsed_read = (double(end-begin)/CLOCKS_PER_SEC);
	
	if (png_get_color_type(png_ptr, info_ptr) == PNG_COLOR_TYPE_RGB)
		abort_("[process_file] input file is PNG_COLOR_TYPE_RGB but must be PNG_COLOR_TYPE_RGBA ",
						"(lacks the alpha channel)");

	if (png_get_color_type(png_ptr, info_ptr) != PNG_COLOR_TYPE_RGBA)
		abort_("[process_file] color_type of input file must be PNG_COLOR_TYPE_RGBA (%d) (is %d)",
					 PNG_COLOR_TYPE_RGBA, png_get_color_type(png_ptr, info_ptr));
	
	N = height;
	  // numero de Threads en cada dimension 
	nThreads = SIZE;

  // numero de Blocks en cada dimension 
	nBlocks = N/nThreads; 
	
	numBytes = N * N * sizeof(char);
	numBytesF = N * N * sizeof(float);
	
	dim3 dimGrid(nBlocks, nBlocks, 1);
	dim3 dimBlock(nThreads, nThreads, 1);
	
	hipEventCreate(&E0);
	hipEventCreate(&E1);
	hipEventCreate(&E2);
	hipEventCreate(&E3);
  	
	if (PINNED) {
	// Obtiene Memoria [pinned] en el host
		hipHostMalloc((unsigned char**)&row_pointers, numBytes); 
		hipHostMalloc((vectorr)&h_out, numBytesF); 
	}
	else {
		// Obtener Memoria en el host
		row_pointers = (unsigned char**) malloc(numBytes); 
		h_out = (vectorr) malloc(numBytesF); 
	}
  
  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
  
  
	// Obtener Memoria en el device
  hipMalloc((unsigned char **)&d_in, numBytes); 
  hipMalloc((vectorr)&d_out, numBytesF); 
  
  // Copiar datos desde el host en el device 
  hipMemcpy((void**)d_in, row_pointers, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_out, h_out, numBytesF, hipMemcpyHostToDevice);
  
  
  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);
  
  fprintf(stderr, "Abans kernel\n");
  // Ejecutar el kernel 
  /***********************/
  /***********************/
  /***********************/
  /***********************/
  img2bw<<<dimGrid, dimBlock>>>(N, N, (unsigned char**) d_in, d_out);
  /***********************/
  /***********************/
  /***********************/
  /***********************/
  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host 
  hipMemcpy(h_out, d_out, numBytes, hipMemcpyDeviceToHost); 

  // Liberar Memoria del device 
  hipFree((void**)d_in);
  hipFree(d_out);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nKERNEL 00\n");
  printf("Dimensiones: %dx%d\n", N, N);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
  if (PINNED) printf("Usando Pinned Memory\n");
         else printf("NO usa Pinned Memory\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
  printf("Rendimiento Global: %4.2f GFLOPS\n", (4.0 * (float) N * (float) N) / (1000000.0 * TiempoTotal));
  printf("Rendimiento Kernel: %4.2f GFLOPS\n", (4.0 * (float) N * (float) N) / (1000000.0 * TiempoKernel));

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

	//~ begin = clock();
	//~ vectorr m = img2bw(width, height, row_pointers);
	//~ end = clock();
	//~ double elapsed_bw = double(end - begin) / CLOCKS_PER_SEC;
	//~ fprintf(stderr, "B&W2 -- %f\n", elapsed_bw);
	//~ vectorr filt;
	//~ vectorr C;
	//~ InitM(3,3,filt);
	//~ gaussFilt(3, filt);
	//~ print(3,3,filt);
	//~ C = TestCM(width, height, 3, m, filt);
	//~ fprintf(stderr, "TEST\n");
	
	row_pointers = greyChar(width, height, h_out);
	fprintf(stderr, "GREYED\n");
	write_png_file(output);	
	
	return 0;
}    

